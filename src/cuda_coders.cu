#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda_coders.h"

#include "g722coder.h"

#define checkCudaErrors(val) CheckErrors( (val), __FILE__, __LINE__)

__device__ short int* q6_ptr;
__device__ short int * misil_ptr;
__device__ short int * coef_qmf_ptr;
__device__ short int * ril4_ptr;
__device__ short int * risil_ptr;
__device__ short int * oq4_ptr;
__device__ short int * wl_ptr;
__device__ short int * ila_ptr;
__device__ short int * misih_ptr;
__device__ short int * ih2_ptr;
__device__ short int * sih_ptr;
__device__ short int * oq2_ptr;
__device__ short int * wh_ptr;


void CheckErrors(hipError_t cuda_error, const char* const file, const int line)
{
	if (cuda_error != hipSuccess)
	{
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(cuda_error) << std::endl;
		exit(1);
	}		
}

__global__ void CudaKernelG711aToPcm(unsigned char* d_alaw_data_ptr, short int* d_pcm_data_ptr)
{
	unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);
	
	short int quantization_value;
	short int quantization_segment;
	unsigned char alaw_data;

	for (int k=0; k<160; k++)
	{
		alaw_data = d_alaw_data_ptr[idx+k];	
		alaw_data^=0x55;

		quantization_value= (alaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)alaw_data & (0x70)) >> (4);
		switch (quantization_segment)
		{
		case 0: 
			quantization_value+=(0x0008);
			break;
		case 1:
			quantization_value+=(0x0108);
			break;
		default:
			quantization_value+=(0x0108);
			quantization_value <<= (quantization_segment-1);
		};

		d_pcm_data_ptr[idx+k]=((alaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

// memory coalesced version of alaw to pcm conversion
__global__ void CudaKernelG711aToPcmCM(unsigned char* d_alaw_data_ptr, short int* d_pcm_data_ptr)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;

	short int quantization_value;
	short int quantization_segment;
	unsigned char alaw_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
		alaw_data = d_alaw_data_ptr[idx];	
		alaw_data^=0x55;

		quantization_value= (alaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)alaw_data & (0x70)) >> (4);
		switch (quantization_segment)
		{
		case 0: 
			quantization_value+=(0x0008);
			break;
		case 1:
			quantization_value+=(0x0108);
			break;
		default:
			quantization_value+=(0x0108);
			quantization_value <<= (quantization_segment-1);
		};

		d_pcm_data_ptr[idx]=((alaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

// memory coalesced version of pcm to alaw conversion
__global__ void CudaKernelPcmToG711aCM(short int* d_pcm_data_ptr, unsigned char* d_alaw_data_ptr)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;

	short int quantization_value;
	short int quantization_segment;
	short int pcm_data;
	unsigned char alaw_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
		alaw_data = 0;
		pcm_data = d_pcm_data_ptr[idx];
		quantization_value=(pcm_data<0) ? ((~pcm_data)>>4) : (pcm_data>>4);
		
		if(quantization_value>15)
		{
			quantization_segment=1;
			while(quantization_value>(16+15))
			{
				quantization_value>>=1;
				quantization_segment++;
			}
			quantization_value-=16;

			alaw_data=quantization_value + (quantization_segment << 4);
		}

		if(pcm_data>=0)
			alaw_data |= 0x80;

		alaw_data^=0x55;
		d_alaw_data_ptr[idx] = alaw_data;
	}
}

__global__ void CudaKernelG711uToPcm(unsigned char* d_ulaw_data_ptr, short int* d_pcm_data_ptr)
{
	unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);

	short int quantization_value;
	short int quantization_segment;
	unsigned char ulaw_data;

	for (int k=0; k<160; k++)
	{
		ulaw_data=~(d_ulaw_data_ptr[idx+k]);

		quantization_value= (ulaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)ulaw_data & (0x70)) >> (4);

		quantization_value += 0x0084;
		quantization_value <<= quantization_segment;

		quantization_value-=(32);

		d_pcm_data_ptr[idx+k]=((ulaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

__global__ void CudaKernelG711uToPcmCM(unsigned char* d_ulaw_data_ptr, short int* d_pcm_data_ptr)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;

	short int quantization_value;
	short int quantization_segment;
	unsigned char ulaw_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
		ulaw_data=~(d_ulaw_data_ptr[idx]);

		quantization_value= (ulaw_data & (0xf)) << 4;
		quantization_segment = ((unsigned)ulaw_data & (0x70)) >> (4);

		quantization_value += 0x0084;
		quantization_value <<= quantization_segment;

		quantization_value-=(32);

		d_pcm_data_ptr[idx]=((ulaw_data & (0x80))?quantization_value:-quantization_value);
	}
}

__global__ void CudaKernelPcmToG711uCM(short int* d_pcm_data_ptr, unsigned char* d_ulaw_data_ptr)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int total_threads = blockDim.x * gridDim.x;
		
	short int quantization_value;
	short int quantization_segment = 1;
	unsigned char ulaw_data;
	short int pcm_data;

	for (int k=0; k<160; k++, idx+=total_threads)
	{
		pcm_data = d_pcm_data_ptr[idx];

		quantization_value=(pcm_data<0) ? (((~pcm_data)>>2)+33) : ((pcm_data>>2)+33);

		if (quantization_value > (0x1FFF))	// clip to 8192
			quantization_value = (0x1FFF);

		quantization_segment = 1;
		// Determination of quantization segment
		for (short int i = (quantization_value >> 6); i; i>>= 1)
			quantization_segment++;

		ulaw_data =  (((0x08 - quantization_segment) << 4) | (0x000F - ((quantization_value  > quantization_segment) & 0x000F)));

		if (pcm_data >= 0)
			ulaw_data |= 0x80;

		d_ulaw_data_ptr[idx] = ulaw_data;
	}
}



/*
__global__ void CudaKernelG722ToPcm(unsigned char* d_g722_data_ptr, short int* d_pcm_data_ptr)
{
	unsigned int idx = 160*(threadIdx.x + blockDim.x * blockIdx.x);
	
	unsigned char g722_data;

	for (int k=0; k<160; k++)
	{
		g722_data=d_g722_data_ptr[idx+k];

		d_pcm_data_ptr[idx+k]=0;
	}
}
*/

__device__ short int CudaConvertLongToShort(int in_value)
{
	if (in_value > 32767)
		return 32767;
	else if (in_value < -32768)
		return -32768;
	else
		return (short)in_value;
}

__global__ void CudaKernelG722ToPcmCM(unsigned char* d_g722_data_ptr, short int* d_pcm_data_ptr, int* d_band_data_ptr, int* d_g722_consts_ptr, unsigned int no_of_data)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	unsigned int total_threads = blockDim.x * gridDim.x;

	unsigned char g722_data;
	
	int number_of_chunks = no_of_data/160;
	if (idx >= number_of_chunks)
		return;

	// pointers for constants, maybe copy these to shared mem
	int* wl_dec = d_g722_consts_ptr;
	int* rl42 = d_g722_consts_ptr + 8;
	int* ilb = rl42 + 16;
	int* qm4 = ilb + 32;
	int* qm6 = qm4 + 16;

	// copy band data to local variables
	int band_s = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_sp = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_sz = d_band_data_ptr[idx];
	idx += number_of_chunks;

	int band_r[3], band_a[3], band_ap[3], band_p[3];
	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
		band_r[k] = d_band_data_ptr[idx];
		band_a[k] = d_band_data_ptr[idx + 3*number_of_chunks];
		band_ap[k] = d_band_data_ptr[idx + 6*number_of_chunks];
		band_p[k] = d_band_data_ptr[idx + 9*number_of_chunks];
	}
	idx += 9*number_of_chunks;

	int band_d[7], band_b[7], band_bp[7], band_sg[7];
	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
		band_d[k] = d_band_data_ptr[idx];
		band_b[k] = d_band_data_ptr[idx + 7*number_of_chunks];
		band_bp[k] = d_band_data_ptr[idx + 14*number_of_chunks];
		band_sg[k] = d_band_data_ptr[idx + 21*number_of_chunks];
	}
	idx += 21*number_of_chunks;

	int band_nb = d_band_data_ptr[idx];
	idx += number_of_chunks;
	int band_det = d_band_data_ptr[idx];
	//band_det=32;

	int dlowt;
	int rlow;
	int wd1;
	int wd2;
	int wd3;

	idx = threadIdx.x + blockDim.x * blockIdx.x;
	for (int k=0; k<160; k++, idx+=number_of_chunks)
	{
		g722_data=d_g722_data_ptr[idx];

		wd1 = g722_data & 0x3F;
		wd2 = qm6[wd1];
		wd1 >>= 2;

		/********************** Block 5 *******************/
		 // INVQBL (ITU page 43), compute quantized difference signal for the decoder output in the lower sub-band
		 wd2 = (band_det * wd2) >> 15;
		 // RECONS ( ITU page 41), compute reconstructed signal for the adaptive predictor
		 rlow = band_s + wd2;

		/********************** Block 6 ********************/
		// LIMIT (ITU page 44), limit the output reconstructed signal
		if (rlow > 16383)
			rlow = 16383;
		else if (rlow < -16384)
			 rlow = -16384;

		/********************** Block 2 ***********************/	
		// INVQAL (ITU page 37), compute the quantized differences signal for the adaptive predictor in the lower sub-band
		wd2 = qm4[wd1];
		dlowt = (band_det * wd2) >> 15;

		/********************** Block 3 ************************/
		// LOGSCL (ITU page 38), update the logarithmic quantizer scale factor in the lower sub-band
		wd2 = rl42[wd1];
		 wd1 = (band_nb * 127) >> 7;
		wd1 += wl_dec[wd2];
		if (wd1 < 0)
			wd1 = 0;
		else if (wd1 > 18432)
			wd1 = 18432;
		band_nb = wd1;

		// SCALEL (ITU page 38), compute the quantizer scale factor in the lower sub-band 
		wd1 = (band_nb >> 6) & 31;
		wd2 = 8 - (band_nb >> 11);
		wd3 = (wd2 < 0)	 ?  (ilb[wd1] << -wd2)	:  (ilb[wd1] >> wd2);
		band_det = wd3 << 2;

		/********************** Block 4 **************************/

		// RECONS (ITU page 41), compute reconstructed signal for the adaptive predictor
		band_d[0] = dlowt;
		 band_r[0] = CudaConvertLongToShort(band_s + dlowt);

		 // PARREC (ITU page 40), compute partially reconstructed signal
		band_p[0] = CudaConvertLongToShort(band_sz + dlowt);

		// UPPOL2 (ITU page 41), update second predictor coefficient
		int i;  // loop variable
		for (i = 0;	 i < 3;	 i++)
			band_sg[i] = band_p[i] >> 15;
		wd1 = CudaConvertLongToShort(band_a[1] << 2);

		wd2 = (band_sg[0] == band_sg[1])	?  -wd1	 :  wd1;
		if (wd2 > 32767)
			wd2 = 32767;
		wd3 = (band_sg[0] == band_sg[2])	?  128	:  -128;
		wd3 += (wd2 >> 7);
		wd3 += (band_a[2]*32512) >> 15;
		if (wd3 > 12288)
			wd3 = 12288;
		else if (wd3 < -12288)
			wd3 = -12288;
		band_ap[2] = wd3;

		// UPPOL1 (ITU page 42), update first predictor coefficient
		band_sg[0] = band_p[0] >> 15;
		band_sg[1] = band_p[1] >> 15;
		wd1 = (band_sg[0] == band_sg[1])	?  192	:  -192;
		wd2 = (band_a[1]*32640) >> 15;

		band_ap[1] = CudaConvertLongToShort(wd1 + wd2);
		 wd3 = CudaConvertLongToShort(15360 - band_ap[2]);
		if (band_ap[1] > wd3)
			band_ap[1] = wd3;
		else if (band_ap[1] < -wd3)
			 band_ap[1] = -wd3;

		// UPZERO (ITU page 41), update sixth order predictor coefficients
		wd1 = (dlowt == 0)  ?  0  :  128;
		band_sg[0] = dlowt >> 15;
		for (i = 1;	 i < 7;	 i++)
		{
			band_sg[i] = band_d[i] >> 15;
			wd2 = (band_sg[i] == band_sg[0])  ?  wd1  :  -wd1;
			wd3 = (band_b[i]*32640) >> 15;
			band_bp[i] = CudaConvertLongToShort(wd2 + wd3);
		}

		// DELAYA (ITU page 38), memory block delay 
		for (i = 6;	 i > 0;	 i--)
		{
			 band_d[i] = band_d[i - 1];
			band_b[i] = band_bp[i];
		}

		for (i = 2;	 i > 0;	 i--)
		{
			band_r[i] = band_r[i - 1];
			band_p[i] = band_p[i - 1];
			band_a[i] = band_ap[i];
		}

		// FILTEP (ITU page 43), compute predictor output signal, poles
		wd1 = CudaConvertLongToShort(band_r[1] + band_r[1]);
		wd1 = (band_a[1]*wd1) >> 15;
		wd2 = CudaConvertLongToShort(band_r[2] + band_r[2]);
		wd2 = (band_a[2]*wd2) >> 15;
		band_sp = CudaConvertLongToShort(wd1 + wd2);

		// FILTEZ (ITU page 42), compute predictor output signal, zeros
		band_sz = 0;
		for (i = 6;	 i > 0;	 i--)
		{
			wd1 = CudaConvertLongToShort(band_d[i] + band_d[i]);
			band_sz += (band_b[i]*wd1) >> 15;
		}
		band_sz = CudaConvertLongToShort(band_sz);

		// PREDIC (ITU page 43), compute predictor output value
		band_s = CudaConvertLongToShort(band_sp + band_sz);

		d_pcm_data_ptr[idx]=(short int)rlow;
	}

	// copy local variables back to band data
	idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	d_band_data_ptr[idx] = band_s;
	idx += number_of_chunks;
	band_sp = d_band_data_ptr[idx]= band_sp;
	idx += number_of_chunks;
	d_band_data_ptr[idx] = band_sz;
	idx += number_of_chunks;
	
	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
		d_band_data_ptr[idx] = band_r[k];
		d_band_data_ptr[idx + 3*number_of_chunks] = band_a[k];
		d_band_data_ptr[idx + 6*number_of_chunks] = band_ap[k];
		d_band_data_ptr[idx + 9*number_of_chunks] = band_p[k];
	}
	idx += 9*number_of_chunks;

	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
		d_band_data_ptr[idx] = band_d[k];
		d_band_data_ptr[idx + 7*number_of_chunks] = band_b[k];
		d_band_data_ptr[idx + 14*number_of_chunks] = band_bp[k];
		d_band_data_ptr[idx + 21*number_of_chunks] = band_sg[k];
	}
	idx += 21*number_of_chunks;

	d_band_data_ptr[idx] = band_nb;
	idx += number_of_chunks;
	d_band_data_ptr[idx] = band_det;
	idx += number_of_chunks;
}

// *******************************************************************************************************



__device__ int CudaSaturateAdd(int op1, int op2)
{
	int out = op1 + op2;
	if ((((op1 ^ op2) & MIN_32) == 0) && ((out ^ op1) & MIN_32))
		out = (op1 < 0) ? MIN_32 : MAX_32;
	return out;
}

__device__ int CudaSaturateSubtract(int op1, int op2)
{
	int out = op1 - op2;
	if ((((op1 ^ op2) & MIN_32) != 0) && ((out ^ op1) & MIN_32))
		out = (op1 < 0L) ? MIN_32 : MAX_32;
	return out;
}

__device__ int CudaShiftRight(int op1, short int op2)
{
	if (op2 > 0)
	{
		if (op2 >= 31)
			return (op1 < 0) ? -1 : 0;
		else
			return op1 >> op2;
	}
	return op1;
}

__device__ int CudaShiftLeft(int op1, short int op2)
{
	if (op2 > 0)
		for (; op2 > 0; op2--)
		{
			if (op1 > 0X3fffffff)
				return MAX_32;
			else if (op1 < (int)0xc0000000)
				return MIN_32;
			op1 *= 2;
		}
	return op1;
}


__device__ short int CudaShiftLeftShort(short int op1, short int op2)
{
	if (op1 > 0)
	{
		int result = ((int)op1) * ((int) 1 << op2);

		if ((op2 > 15 && op1 != 0) || (result != (int) ((short int) result)))
			return (op1 > 0) ? MAX_16 : MIN_16;
		else
			return (short int)result;
	}
	return op1;
}

__device__ short int CudaShiftRightShort(short int op1, short int op2)
{
	if (op2 > 0)
	{
		if (op2 >= 15)
			return (op1 < 0) ? -1 : 0;
		else
			if (op1 < 0)
				return ~((~op1) >> op2);
			else
				return op1 >> op2;
	}
	return op1;
}

__device__ int CudaClamp15ToBits(int op)
{
	if (op > 16383)
		return 16383;
	else if (op < -16384)
		return -16384;
	return op;
}

__device__ int CudaMultiplyAdd(int add_op, short int mul_op1, short int mul_op2)
{
	return CudaSaturateAdd(add_op, ((int)mul_op1 * (int)mul_op2));
}

__device__ short int CudaSaturate(int op)
{
	if (op > MAX_16)
		return MAX_16;
	else if (op < MIN_16)
		return MIN_16;
	return op;
}

__device__ short int CudaSaturateSubtractShort(short int op1, short int op2)
{
	return CudaSaturate (((int)op1 - op2));
}

__device__ short int CudaSaturateAddShort(short int op1, short int op2)
{
	return CudaSaturate (((int)op1 + op2));
}

__device__ short int CudaScaledMult(short int op1, short int op2)
{
	int product = (((int)op1 * (int)op2) & (int)(0xffff8000)) >> 15;

	if (product & (int)0x00010000)
		product |= (int)0xffff0000;

	return (short int)product;
}

__device__ short int CudaQuantl(short int el, short int detl)
{
	short int sil = CudaShiftRightShort(el, 15);
	short int wd = CudaSaturateSubtractShort(MAX_16,(el & MAX_16));
	short int mil = 0;

	if (sil == 0)
		wd = el;

	short int val = CudaScaledMult(CudaShiftLeftShort(q6_ptr[mil], 3), detl);
	while (CudaSaturateSubtractShort(val,wd) <= 0)
	{
		if (CudaSaturateSubtractShort(mil, 30) == 0)
			break;
		else
		{
			mil = CudaSaturateAddShort(mil, 1);
			val = CudaScaledMult(CudaShiftLeftShort(q6_ptr[mil], 3), detl);
		}
	}

	sil = CudaSaturateAddShort(sil, 1);

	return misil_ptr[sil*32 + mil];
}

__device__ short int CudaQuanth(short int eh, short int deth)
{
	short int sih = CudaShiftRightShort(eh, 15);
	short int wd = CudaSaturateSubtractShort(MAX_16, (eh & MAX_16));

	if (sih == 0)
		wd = eh;

	short int mih = 1;

	if (CudaSaturateSubtractShort(wd, CudaScaledMult(CudaShiftLeftShort(564, 3), deth)) >= 0)
		mih = 2;

	sih = CudaSaturateAddShort(sih, 1);

	return misih_ptr[sih*3 + mih];
}

__device__ short int CudaInvqal(short int il, short int detl)
{
	short int ril = CudaShiftRightShort(il, 2);
	short int wd1 = CudaShiftLeftShort(oq4_ptr[ril4_ptr[ril]], 3);
	short int wd2 = -wd1;

	if (risil_ptr[ril] == 0)
		wd2 = wd1;

	return CudaScaledMult(detl, wd2);
}

__device__ short int CudaInvqah(short int ih, short int deth)
{
	short int wd1 = CudaShiftLeftShort(oq2_ptr[ih2_ptr[ih]], 3);
	short int wd2 = -wd1;

	if (sih_ptr[ih] == 0)
		wd2 = wd1;

	return CudaScaledMult(wd2, deth);
}

__device__ short int CudaLogscl(short int il, short int nbl)
{
	short int ril = CudaShiftRightShort(il, 2);
	short int wd = CudaScaledMult(nbl, 32512);
	short int il4 = ril4_ptr[ril];
	short int nbpl = CudaSaturateAddShort (wd, wl_ptr[il4]);

	if (nbpl < 0)
		nbpl = 0;

	if (CudaSaturateSubtractShort(nbpl, 18432) > 0)
		nbpl = 18432;

	return nbpl;
}

__device__ short int CudaLogsch(short int ih, short int nbh)
{
	short int wd = CudaScaledMult(nbh, 32512);
	short int nbph = CudaSaturateAddShort(wd, wh_ptr[ih2_ptr[ih]]);

	if(nbph < 0)
		nbph = 0;

	if(CudaSaturateSubtractShort(nbph, 22528) > 0)
		nbph = 22528;

	return nbph;
}

__device__ short int CudaScalel(short int nbpl)
{
	short int wd1 = CudaShiftRightShort(nbpl, 6) & 511;
	short int wd2 = CudaSaturateAddShort(wd1, 64);
	return (CudaShiftLeftShort(CudaSaturateAddShort(ila_ptr[wd2], 1), 2));
}

__device__ short int CudaScaleh(short int nbph)
{
	short int wd = CudaShiftRightShort(nbph, 6) & 511;
	return CudaShiftLeftShort(CudaSaturateAddShort(ila_ptr[wd], 1), 2);
}


__device__ void CudaUpzero(short int* dlt_ptr, short int* bl_ptr)
{
	short int wd1 = 128;

	if (dlt_ptr[0] == 0)
		 wd1 = 0;

	short int sg0 = CudaShiftRightShort(dlt_ptr[0], 15);

	for (short int i = 6; i > 0; i--)
	{
		short int wd2 = CudaSaturateSubtractShort (0, wd1);
		if(sg0 == CudaShiftRightShort(dlt_ptr[i], 15))
			wd2 = CudaSaturateAddShort (0, wd1);

		bl_ptr[i] = CudaSaturateAddShort(wd2, CudaScaledMult(bl_ptr[i], 32640));
		dlt_ptr[i] = dlt_ptr[i - 1];
	}
}

__device__ void CudaUppol1(short int* al_ptr, short int* plt_ptr)
{
	short int sg0 = CudaShiftRightShort(plt_ptr[0], 15);
	short int sg1 = CudaShiftRightShort(plt_ptr[1], 15);
	short int wd1 = -192;

	if (CudaSaturateSubtractShort(sg0, sg1) == 0)
		wd1 = 192;

	short int wd2 = CudaScaledMult (al_ptr[1], 32640);
	short int apl1 = CudaSaturateAddShort(wd1, wd2);
	short int wd3 = CudaSaturateSubtractShort(15360, al_ptr[2]);

	if (CudaSaturateSubtractShort(apl1, wd3) > 0)
		apl1 = wd3;
	else if (CudaSaturateAddShort(apl1, wd3) < 0)
		apl1 = -wd3;

	/* Shift of the plt signals */
	plt_ptr[2] = plt_ptr[1];
	plt_ptr[1] = plt_ptr[0];
	al_ptr[1] = apl1;
}

__device__ void CudaUppol2(short int* al_ptr, short int* plt_ptr)
{
	short int sg0 = CudaShiftRightShort(plt_ptr[0], 15);
	short int sg1 = CudaShiftRightShort(plt_ptr[1], 15);
	short int sg2 = CudaShiftRightShort(plt_ptr[2], 15);
	short int wd1 = CudaShiftLeftShort(al_ptr[1], 2);
	short int wd2 = CudaSaturateAddShort(0, wd1);

	if (CudaSaturateSubtractShort(sg0, sg1) == 0)
		wd2 = CudaSaturateSubtractShort(0, wd1);

	wd2 = CudaShiftRightShort(wd2, 7);
	short int wd3 = -128;

	if (CudaSaturateSubtractShort(sg0, sg2) == 0)
		wd3 = 128;

	short int wd4 = CudaSaturateAddShort (wd2, wd3);
	short int wd5 = CudaScaledMult(al_ptr[2], 32512);
	short int apl2 = CudaSaturateAddShort(wd4, wd5);

	if (CudaSaturateSubtractShort(apl2, 12288) > 0)
		apl2 = 12288;

	if (CudaSaturateSubtractShort(apl2, -12288) < 0)
		apl2 = -12288;

	al_ptr[2] = apl2;
}

__device__ short int CudaFiltez(short int* dlt_ptr, short int* bl_ptr)
{
	short int szl = 0;

	for (short int i = 6; i > 0; i--)
	{
		short int wd = CudaSaturateAddShort(dlt_ptr[i], dlt_ptr[i]);
		wd = CudaScaledMult(wd, bl_ptr[i]);
		szl = CudaSaturateAddShort(szl, wd);
	}
	return szl;
}

__device__ short int CudaFiltep(short int* rlt_ptr, short int* al_ptr)
{
	// shift of rlt
	rlt_ptr[2] = rlt_ptr[1];		
	rlt_ptr[1] = rlt_ptr[0];		

	short int wd1 = CudaSaturateAddShort(rlt_ptr[1], rlt_ptr[1]);
	wd1 = CudaScaledMult(al_ptr[1], wd1);
	short int wd2 = CudaSaturateAddShort(rlt_ptr[2], rlt_ptr[2]);
	wd2 = CudaScaledMult(al_ptr[2], wd2);
	return CudaSaturateAddShort(wd1, wd2);
}

__device__ void CudaQmfTx(short int xin0, short int xin1, short int& xl, short int& xh, short int* band_qmf_tx_delayx)
{
	int accuma;
	int accumb;
	int comp_low;
	int comp_high;

	const short int* pcoef = coef_qmf_ptr;
	short int* pdelayx = band_qmf_tx_delayx;

	/* Saving past samples in delay line */
	band_qmf_tx_delayx[1] = xin1;
	band_qmf_tx_delayx[0] = xin0;

	accuma = (int)*pcoef++, (int)*pdelayx++;
	accumb = (int)*pcoef++, (int)*pdelayx++;

	for(short int i = 1; i < 12; i++)
	{
		accuma = CudaMultiplyAdd(accuma, *pcoef++, *pdelayx++);
		accumb = CudaMultiplyAdd(accumb, *pcoef++, *pdelayx++);
	}

	/* Descaling and shift of the delay line */
	for (short int i = 0; i < 22; i++)
		band_qmf_tx_delayx[23 - i] = band_qmf_tx_delayx[21 - i];

	comp_low = CudaSaturateAdd (accuma, accumb);
	comp_low = CudaSaturateAdd (comp_low, comp_low);
	comp_high = CudaSaturateSubtract (accuma, accumb);
	comp_high = CudaSaturateAdd (comp_high, comp_high);
	xl = CudaClamp15ToBits (CudaShiftRight(comp_low, 16));
	xh = CudaClamp15ToBits (CudaShiftRight(comp_high, 16));
}

__device__ short int CudaLsbCod(short int xl, short int* band_dlt, short int* band_plt, short int* band_rlt, 
		short int& band_sl, short int& band_detl, short int& band_nbl, short int& band_szl, short int* band_bl, 
		short int* band_al, short int& band_spl)
{
	short int il = CudaQuantl (CudaSaturateSubtractShort (xl, band_sl), band_detl);
	band_dlt[0] = CudaInvqal (il, band_detl);
	short int nbpl = CudaLogscl (il, band_nbl);
	band_nbl = nbpl;
	band_detl = CudaScalel (nbpl);
	band_plt[0] = CudaSaturateAddShort (band_dlt[0], band_szl);   /* parrec */
	band_rlt[0] = CudaSaturateAddShort (band_sl, band_dlt[0]);    /* recons */
	CudaUpzero (band_dlt, band_bl);
	CudaUppol2 (band_al, band_plt);
	CudaUppol1 (band_al, band_plt);
	band_szl = CudaFiltez(band_dlt, band_bl);
	band_spl = CudaFiltep(band_rlt, band_al);
	band_sl = CudaSaturateAddShort (band_spl, band_szl);          /* predic */

	/* Return encoded sample */
	return il;
}

__device__ short int CudaHsbCod(short int xh, short int* band_dh, short int* band_ph, short int* band_rh,
			short int& band_sh, short int& band_deth, short int& band_nbh, short int& band_szh,
			short int* band_bh, short int* band_ah, short int& band_sph)
{
	short int ih = CudaQuanth (CudaSaturateSubtractShort(xh, band_sh), band_deth);
	band_dh[0] = CudaInvqah (ih, band_deth);
	short int nbph = CudaLogsch (ih, band_nbh);
	band_nbh = nbph;
	band_deth = CudaScaleh (nbph);
	band_ph[0] = CudaSaturateAddShort(band_dh[0], band_szh);   /* parrec */
	band_rh[0] = CudaSaturateAddShort(band_sh, band_dh[0]);    /* recons */
	CudaUpzero (band_dh, band_bh);
	CudaUppol2 (band_ah, band_ph);
	CudaUppol1 (band_ah, band_ph);
	band_szh = CudaFiltez (band_dh, band_bh);
	band_sph = CudaFiltep (band_rh, band_ah);
	band_sh = CudaSaturateAddShort(band_sph, band_szh);        /* predic */

	return ih;
}


// ******************************************************************************************************

__global__ void CudaKernelPcmToG722CM(short int* d_pcm_data_ptr, unsigned char* d_g722_data_ptr, short int* d_band_data_ptr, 
short int* d_g722_consts_ptr, unsigned int no_of_data)
{
	unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
//	unsigned int total_threads = blockDim.x * gridDim.x;

	int number_of_chunks = no_of_data/320;
	if (idx >= number_of_chunks)
		return;

	// pointers for constants, maybe copy these to shared mem
	coef_qmf_ptr = d_g722_consts_ptr;
	misil_ptr = d_g722_consts_ptr + 24;
	q6_ptr = d_g722_consts_ptr + 88;
	ril4_ptr = d_g722_consts_ptr + 119;
	risil_ptr = d_g722_consts_ptr + 135;
	oq4_ptr = d_g722_consts_ptr + 151;
	wl_ptr = d_g722_consts_ptr + 159;
	ila_ptr = d_g722_consts_ptr + 167;
	misih_ptr = d_g722_consts_ptr + 520;
	ih2_ptr = d_g722_consts_ptr + 526;
	sih_ptr = d_g722_consts_ptr + 530;
	oq2_ptr = d_g722_consts_ptr + 534;
	wh_ptr = d_g722_consts_ptr + 537;

	// copy band data to local variables
	// detl
	short int band_detl = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// deth
	short int band_deth = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// nbl
	short int band_nbl = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// sl
	short int band_sl = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// spl
	short int band_spl = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// szl
	short int band_szl = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// nbh
	short int band_nbh = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// sh
	short int band_sh = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// sph
	short int band_sph = d_band_data_ptr[idx];
	idx += number_of_chunks;
	// szh
	short int band_szh = d_band_data_ptr[idx];
	idx += number_of_chunks;


	short int band_al[3], band_plt[3], band_rlt[3], band_ah[3], band_ph[3], band_rh[3];
	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
		band_al[k] = d_band_data_ptr[idx];
		band_plt[k] = d_band_data_ptr[idx + 3*number_of_chunks];
		band_rlt[k] = d_band_data_ptr[idx + 6*number_of_chunks];
		band_ah[k] = d_band_data_ptr[idx + 9*number_of_chunks];
		band_ph[k] = d_band_data_ptr[idx + 12*number_of_chunks];
		band_rh[k] = d_band_data_ptr[idx + 15*number_of_chunks];
	}
	idx += 15*number_of_chunks;

	short int band_bl[7], band_dlt[7], band_bh[7], band_dh[7];
	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
		band_bl[k] = d_band_data_ptr[idx];
		band_dlt[k] = d_band_data_ptr[idx + 7*number_of_chunks];
		band_bh[k] = d_band_data_ptr[idx + 14*number_of_chunks];
		band_dh[k] = d_band_data_ptr[idx + 21*number_of_chunks];
	}
	idx += 21*number_of_chunks;

	short int band_qmf_tx_delayx[24], band_qmf_rx_delayx[24];
	for(int k=0; k<24; k++, idx+=number_of_chunks)
	{
		band_qmf_tx_delayx[k] = d_band_data_ptr[idx];
		band_qmf_rx_delayx[k] = d_band_data_ptr[idx + 24*number_of_chunks];
	}
	idx += 24*number_of_chunks;

	short int xin1;
	short int xin0;
	short int xl;
	short int il;
	short int xh;
	short int ih;

	idx = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int idx1 = threadIdx.x + blockDim.x * blockIdx.x;
	for (int k=0; k<160; k++, idx += number_of_chunks, idx1 +=(2*number_of_chunks))
	{
		xin1 = d_pcm_data_ptr[idx1];
		xin0 = d_pcm_data_ptr[idx1 + number_of_chunks];

		// Calculation of the synthesis QMF samples 
		// qmf_tx (xin0, xin1, &xl, &xh, encoder);
		CudaQmfTx(xin0, xin1, xl, xh, band_qmf_tx_delayx);

		// Call the upper and lower band ADPCM encoders
		// il = lsbcod (xl, 0, encoder);
		il = CudaLsbCod(xl, band_dlt, band_plt, band_rlt, band_sl, band_detl, band_nbl, band_szl, band_bl, band_al, band_spl);
		// ih = hsbcod (xh, 0, encoder);
		ih = CudaHsbCod(xh, band_dh, band_ph, band_rh, band_sh, band_deth, band_nbh, band_szh, band_bh, band_ah, band_sph);

		// Mount the output G722 codeword: bits 0 to 5 are the lower-band
		// portion of the encoding, and bits 6 and 7 are the upper-band
		// portion of the encoding 
		// code[i] = s_and(add(shl(ih, 6), il), 0xFF);
		d_g722_data_ptr[idx] = (unsigned char) CudaSaturateAddShort(CudaShiftLeftShort(ih, 6), il);

//		d_g722_data_ptr[idx] = (unsigned char) ((xl + xh) / 2);
	}

	// copy local variables back to band data
	idx = threadIdx.x + blockDim.x * blockIdx.x;
	
	// detl	
	d_band_data_ptr[idx] = band_detl;
	idx += number_of_chunks;
	// deth
	d_band_data_ptr[idx] = band_deth;
	idx += number_of_chunks;
	// nbl
	d_band_data_ptr[idx]= band_nbl;
	idx += number_of_chunks;
	// sl	
	d_band_data_ptr[idx] = band_sl;
	idx += number_of_chunks;
	// spl
	d_band_data_ptr[idx]= band_spl;
	idx += number_of_chunks;
	// szl
	d_band_data_ptr[idx] = band_szl;
	idx += number_of_chunks;
	// nbh
	d_band_data_ptr[idx] = band_nbh;
	idx += number_of_chunks;
	// sh	
	d_band_data_ptr[idx] = band_sh;
	idx += number_of_chunks;
	// sph
	d_band_data_ptr[idx]= band_sph;
	idx += number_of_chunks;
	// szh
	d_band_data_ptr[idx] = band_szh;
	idx += number_of_chunks;

	for(int k=0; k<3; k++, idx+=number_of_chunks)
	{
		d_band_data_ptr[idx] = band_al[k];
		d_band_data_ptr[idx + 3*number_of_chunks] = band_plt[k];
		d_band_data_ptr[idx + 6*number_of_chunks] = band_rlt[k];
		d_band_data_ptr[idx + 9*number_of_chunks] = band_ah[k];
		d_band_data_ptr[idx + 6*number_of_chunks] = band_ph[k];
		d_band_data_ptr[idx + 9*number_of_chunks] = band_rh[k];
	}
	idx += 15*number_of_chunks;

	for(int k=0; k<7; k++, idx+=number_of_chunks)
	{
		d_band_data_ptr[idx] = band_bl[k];
		d_band_data_ptr[idx + 7*number_of_chunks] = band_dlt[k];
		d_band_data_ptr[idx + 14*number_of_chunks] = band_bh[k];
		d_band_data_ptr[idx + 21*number_of_chunks] = band_dh[k];
	}
	idx += 21*number_of_chunks;

	for(int k=0; k<24; k++, idx+=number_of_chunks)
	{
		d_band_data_ptr[idx] = band_qmf_tx_delayx[k];
		d_band_data_ptr[idx + 24*number_of_chunks] = band_qmf_rx_delayx[k];
	}
	idx += 24*number_of_chunks;
}

__global__ void CudaKernelSinusoidalSynthesis(const float* parameter_ptr, int max_sin, int number_of_packets, int samples_per_packet, short int* output_ptr)
{
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= number_of_packets)
        return;

    int size_of_each_row = 6 * number_of_packets;

    float c = 0;
    float d = 0;


        // initialize output ptr
        for (int l = 1; l <= samples_per_packet; l++)
            output_ptr[(l-1)*number_of_packets + idx] = 0;

        // each packet has max_sin sinusoidals to process
        for (int i = 0; i < max_sin; i++)
        {
            // get necessary data
            int offset = i*size_of_each_row + idx*6;
            float Ap = parameter_ptr[offset];
            float  An = parameter_ptr[offset+1];
            float Op = parameter_ptr[offset+2];
            float On = parameter_ptr[offset+3];
            float Wn = parameter_ptr[offset+4];
            float Wp = parameter_ptr[offset+5];
            float epsilon = 0;

            // classify operation
            if (Ap && An)    // interpolation
            {
                // calculate epsilon
                float total = Op - On + (Wp + Wn)*(samples_per_packet/2);
                float var_n = -2*(PI) - total;
                float var_o = -total;
                float var_p = 2*(PI) - total;
                if ((var_n <= var_o) && ((var_o <= var_p) || (var_n <= var_p)))
                {
                    // search through negative side
                    epsilon = var_n;
                    for (int k = 2; k < 100; k++)
                    {
                        if ((-2*PI*k - total) >= epsilon)
                            break;
                        else
                            epsilon = (-2*PI*k - total);
                    }
                }
                else if (((var_n >= var_o) || (var_n >= var_p)) && (var_o >= var_p))
                {
                    // search through positive side
                    epsilon = var_p;
                    for (int k = 2; k < 100; k++)
                    {
                        if (2*PI*k - total >= epsilon)
                            break;
                        else
                            epsilon = 2*PI*k - total;
                    }
              
                }
                else
                {
                    // ((var_n >= var_o) && (var_o <= var_p))
                    epsilon = var_o;
                }

                // calculate coefficients
                c = (Wn - Wp)/(2*samples_per_packet) + 3*epsilon/(samples_per_packet * samples_per_packet);
                d = -2*epsilon/(samples_per_packet*samples_per_packet*samples_per_packet);
                
            }
            else if (An)    // birth
            {
                Wp = Wn;
                Op = On - (Wp * samples_per_packet);
            }
            else if (Ap)    // death
            {
                
            }
            else    // null
            {
                continue;
            }
           
            for (int l = 1; l <= samples_per_packet; l++)
                output_ptr[(l-1)*number_of_packets + idx] += (((An - Ap)*l/samples_per_packet + Ap)*cos(Op + Wp*l + c*l*l + d*l*l*l))*32000;
        }

}


int CudaG711aToPcm(const unsigned char* alaw_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);

	unsigned int size_of_alaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_alaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);

	unsigned char* d_alaw_data_ptr = NULL;
	hipMalloc((void**)&d_alaw_data_ptr, size_of_d_alaw_data);
	checkCudaErrors(hipGetLastError());

	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_alaw_data_ptr, alaw_data_ptr, size_of_alaw_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());

	// launch kernel here
	CudaKernelG711aToPcmCM <<< grid_dim, block_dim >>> (d_alaw_data_ptr, d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipFree(d_alaw_data_ptr);
	checkCudaErrors(hipGetLastError());
   
	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

int CudaG711uToPcm(const unsigned char* ulaw_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);

	unsigned int size_of_ulaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_ulaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);

	unsigned char* d_ulaw_data_ptr = NULL;
	hipMalloc((void**)&d_ulaw_data_ptr, size_of_d_ulaw_data);
	checkCudaErrors(hipGetLastError());

	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_ulaw_data_ptr, ulaw_data_ptr, size_of_ulaw_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());

	// launch kernel here
	CudaKernelG711uToPcmCM <<< grid_dim, block_dim >>> (d_ulaw_data_ptr, d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipFree(d_ulaw_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

int CudaG722ToPcm(const unsigned char* g722_data_ptr, int* band_data_ptr, unsigned int no_of_data, short int* pcm_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);

	unsigned int size_of_g722_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_g722_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);

	unsigned char* d_g722_data_ptr = NULL;
	hipMalloc((void**)&d_g722_data_ptr, size_of_d_g722_data);
	checkCudaErrors(hipGetLastError());

	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_g722_data_ptr, g722_data_ptr, size_of_g722_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());

	// calculate space for band, 45 integers per thread
	unsigned int number_of_d_band_data = grid_dim.x * block_dim.x;
	unsigned int size_of_d_band_data = number_of_d_band_data * 45 * sizeof(int);
	unsigned int number_of_band_data = no_of_data/160;
	unsigned int size_of_band_data = number_of_band_data * 45 * sizeof(int);
	
	//std::cout << "size of band data " << size_of_d_band_data << std::endl;
	
	int* d_band_data_ptr = NULL;
	hipMalloc((void**)&d_band_data_ptr, size_of_d_band_data);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_band_data_ptr, band_data_ptr, size_of_band_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	

	unsigned int size_of_d_g722_consts = sizeof(g722_consts);
	int* d_g722_consts_ptr = NULL;
	hipMalloc((void**)&d_g722_consts_ptr, size_of_d_g722_consts);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_g722_consts_ptr, g722_consts, size_of_d_g722_consts, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	
 

	// launch kernel here
	CudaKernelG722ToPcmCM <<< grid_dim, block_dim >>> (d_g722_data_ptr, d_pcm_data_ptr, d_band_data_ptr, d_g722_consts_ptr, no_of_data);
	checkCudaErrors(hipGetLastError());

	//std::cout << "host pcm data size   : " << size_of_pcm_data << std::endl;
	//std::cout << "device pcm data size : " << size_of_d_pcm_data << std::endl;
	//std::cout << "no of data           : " << no_of_data << std::endl;
	//std::cout << "number of threads    : " << no_of_d_data << std::endl;

	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(band_data_ptr, d_band_data_ptr, size_of_band_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipFree(d_g722_consts_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_band_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_g722_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

int CudaPcmToG711a(const short int* pcm_data_ptr, unsigned int no_of_data, unsigned char* alaw_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);
	
	unsigned int size_of_alaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);
	
	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_alaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);
	
	unsigned char* d_alaw_data_ptr = NULL;
	hipMalloc((void**)&d_alaw_data_ptr, size_of_d_alaw_data);
	checkCudaErrors(hipGetLastError());
	
	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());
	
	hipMemcpy(d_pcm_data_ptr, pcm_data_ptr, size_of_pcm_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(d_alaw_data_ptr, alaw_data_ptr, size_of_alaw_data, hipMemcpyHostToDevice);
//	checkCudaErrors(hipGetLastError());
	
	// launch kernel here
	CudaKernelPcmToG711aCM <<< grid_dim, block_dim >>> (d_pcm_data_ptr, d_alaw_data_ptr);
	checkCudaErrors(hipGetLastError());
//	CudaKernelG711aToPcmCM <<< grid_dim, block_dim >>> (d_alaw_data_ptr, d_pcm_data_ptr);
// 	checkCudaErrors(hipGetLastError());
	
	hipMemcpy(alaw_data_ptr, d_alaw_data_ptr, size_of_alaw_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
//	checkCudaErrors(hipGetLastError());
	
	hipFree(d_alaw_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

int CudaPcmToG711u(const short int* pcm_data_ptr, unsigned int no_of_data, unsigned char* ulaw_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 160))), 1, 1);

	unsigned int size_of_ulaw_data = no_of_data * sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 160;
	unsigned int size_of_d_ulaw_data = no_of_d_data * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);

	unsigned char* d_ulaw_data_ptr = NULL;
	hipMalloc((void**)&d_ulaw_data_ptr, size_of_d_ulaw_data);
	checkCudaErrors(hipGetLastError());

	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_pcm_data_ptr, pcm_data_ptr, size_of_pcm_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(d_ulaw_data_ptr, ulaw_data_ptr, size_of_ulaw_data, hipMemcpyHostToDevice);
//	checkCudaErrors(hipGetLastError());

	// launch kernel here
	CudaKernelPcmToG711uCM <<< grid_dim, block_dim >>> (d_pcm_data_ptr, d_ulaw_data_ptr);
	checkCudaErrors(hipGetLastError());
//	CudaKernelG711uToPcmCM <<< grid_dim, block_dim >>> (d_ulaw_data_ptr, d_pcm_data_ptr);
//	checkCudaErrors(hipGetLastError());

	hipMemcpy(ulaw_data_ptr, d_ulaw_data_ptr, size_of_ulaw_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
//	checkCudaErrors(hipGetLastError());

	hipFree(d_ulaw_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

int CudaPcmToG722(const short int* pcm_data_ptr, short int* band_data_ptr, unsigned int no_of_data, unsigned char* g722_data_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(no_of_data/((float)(block_dim.x * 320))), 1, 1);

	unsigned int size_of_g722_data = (no_of_data / 2)* sizeof(unsigned char);
	unsigned int size_of_pcm_data = no_of_data * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x * 320;
	unsigned int size_of_d_g722_data = (no_of_d_data / 2) * sizeof(unsigned char);
	unsigned int size_of_d_pcm_data = no_of_d_data * sizeof(short int);

	unsigned char* d_g722_data_ptr = NULL;
	hipMalloc((void**)&d_g722_data_ptr, size_of_d_g722_data);
	checkCudaErrors(hipGetLastError());

	short int* d_pcm_data_ptr = NULL;
	hipMalloc((void**)&d_pcm_data_ptr, size_of_d_pcm_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_pcm_data_ptr, pcm_data_ptr, size_of_pcm_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(d_g722_data_ptr, g722_data_ptr, size_of_g722_data, hipMemcpyHostToDevice);
//	checkCudaErrors(hipGetLastError());

	// calculate space for band, 104 short integers per thread
	unsigned int number_of_d_band_data = grid_dim.x * block_dim.x;
	unsigned int size_of_d_band_data = number_of_d_band_data * 104 * sizeof(short int);
	unsigned int number_of_band_data = no_of_data/360;
	unsigned int size_of_band_data = number_of_band_data * 104 * sizeof(short int);
	
	//std::cout << "size of band data " << size_of_d_band_data << std::endl;
	
	short int* d_band_data_ptr = NULL;
	hipMalloc((void**)&d_band_data_ptr, size_of_d_band_data);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_band_data_ptr, band_data_ptr, size_of_band_data, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	

	unsigned int size_of_d_g722_consts = sizeof(full_g722_consts);
	short int* d_g722_consts_ptr = NULL;
	hipMalloc((void**)&d_g722_consts_ptr, size_of_d_g722_consts);
	checkCudaErrors(hipGetLastError());
	hipMemcpy(d_g722_consts_ptr, full_g722_consts, size_of_d_g722_consts, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());	
 

	// launch kernel here
	CudaKernelPcmToG722CM <<< grid_dim, block_dim >>> (d_pcm_data_ptr, d_g722_data_ptr, d_band_data_ptr, d_g722_consts_ptr, no_of_data);
	checkCudaErrors(hipGetLastError());

	//std::cout << "host pcm data size   : " << size_of_pcm_data << std::endl;
	//std::cout << "device pcm data size : " << size_of_d_pcm_data << std::endl;
	//std::cout << "no of data           : " << no_of_data << std::endl;
	//std::cout << "number of threads    : " << no_of_d_data << std::endl;


	hipMemcpy(g722_data_ptr, d_g722_data_ptr, size_of_g722_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
//	hipMemcpy(pcm_data_ptr, d_pcm_data_ptr, size_of_pcm_data, hipMemcpyDeviceToHost);
//	checkCudaErrors(hipGetLastError());

	hipMemcpy(band_data_ptr, d_band_data_ptr, size_of_band_data, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipFree(d_g722_consts_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_band_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_g722_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_pcm_data_ptr);
	checkCudaErrors(hipGetLastError());	

	return 0;
}

void CudaSinusoidalSynthesis(const float* parameter_ptr, int max_sin, int number_of_packets, int samples_per_packet, short int* output_ptr)
{
	dim3 block_dim(THREAD_PER_BLOCK, 1, 1);
	dim3 grid_dim(ceil(number_of_packets/((float)(block_dim.x))), 1, 1);

	unsigned int size_of_parameter_ptr = (number_of_packets * max_sin * 6)* sizeof(float);
	unsigned int size_of_output_ptr = number_of_packets * samples_per_packet * sizeof(short int);

	unsigned int no_of_d_data = grid_dim.x * block_dim.x;
	unsigned int size_of_d_parameter_data = (no_of_d_data ) * sizeof(float) * max_sin * 6;
	unsigned int size_of_d_output_data = no_of_d_data * sizeof(short int) * samples_per_packet;

	float* d_parameter_data_ptr = NULL;
	hipMalloc((void**)&d_parameter_data_ptr, size_of_d_parameter_data);
	checkCudaErrors(hipGetLastError());

	short int* d_output_data_ptr = NULL;
	hipMalloc((void**)&d_output_data_ptr, size_of_d_output_data);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(d_parameter_data_ptr, parameter_ptr, size_of_parameter_ptr, hipMemcpyHostToDevice);
	checkCudaErrors(hipGetLastError());

	// launch kernel here
	CudaKernelSinusoidalSynthesis <<< grid_dim, block_dim >>> (d_parameter_data_ptr, max_sin, number_of_packets, samples_per_packet, d_output_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipMemcpy(output_ptr, d_output_data_ptr, size_of_output_ptr, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());

	hipFree(d_parameter_data_ptr);
	checkCudaErrors(hipGetLastError());

	hipFree(d_output_data_ptr);
	checkCudaErrors(hipGetLastError());
}

void CudaGpuInitialize()
{
    unsigned int size_of_d_dummy_data = 1000000;
    unsigned char* d_dummy_data_ptr = NULL;

    //unsigned int device_count = hipGetDeviceCount();
    //checkCudaErrors(hipGetLastError()); 
    
    //std::cout << "device count : " << device_count << std::endl
	
    hipMalloc((void**)&d_dummy_data_ptr, size_of_d_dummy_data);
    checkCudaErrors(hipGetLastError());
	
    hipMemset((void*)d_dummy_data_ptr, 0, size_of_d_dummy_data);
    checkCudaErrors(hipGetLastError());
	
    hipFree(d_dummy_data_ptr);
    checkCudaErrors(hipGetLastError());
}

bool DetectCudaDevice()
{
    int number_of_cuda_devices = 0;
    std::vector<hipDeviceProp_t> cuda_device_list;

    hipGetDeviceCount(&number_of_cuda_devices);

    // reset the device list
    cuda_device_list.clear();
    for (int i = 0; i < number_of_cuda_devices; i++)
    {
        hipDeviceProp_t cuda_device;
        hipGetDeviceProperties(&cuda_device, i);
        cuda_device_list.push_back(cuda_device);

        std::cout << "Device : " << i << " " << cuda_device.name << std::endl;
        std::cout << "Compute number of device  : " << cuda_device.major << "." << cuda_device.minor << std::endl;
        std::cout << "Concurrent kernels        : " << cuda_device.concurrentKernels << std::endl;
        std::cout << "Number of sm              : " << cuda_device.multiProcessorCount << std::endl;
        std::cout << "Maximum threads per block : " << cuda_device.maxThreadsPerBlock << std::endl;
        std::cout << "Maximum threads per sm    : " << cuda_device.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "Memory pitch              : " << cuda_device.memPitch << std::endl;
        std::cout << "Registers per block       : " << cuda_device.regsPerBlock << std::endl;
        std::cout << "Shared memory per block   : " << cuda_device.sharedMemPerBlock << std::endl;
        std::cout << std::endl;
    }

    return ((number_of_cuda_devices) ? true : false);
}
